#include "hip/hip_runtime.h"
#include <Logger.h>
#include <hip/hip_vector_types.h>

#include <fluids.cuh>

__device__ float2 *m_velocity[2];
__device__ size_t m_velocity_pitch[2];
__device__ float *m_pressure[2];
__device__ size_t m_pressure_pitch[2];
__device__ float *m_divergence;
__device__ size_t m_divergence_pitch;
__device__ float3 *m_dye[2];
__device__ size_t m_dye_pitch[2];

float2 *h_velocity[2];
size_t h_velocity_pitch[2];
float *h_pressure[2];
size_t h_pressure_pitch[2];
float *h_divergence;
size_t h_divergence_pitch;
float3 *h_dye[2];
size_t h_dye_pitch[2];

__global__ void swap_velocity() {
    float2 *tmp_velocity = m_velocity[0];
    m_velocity[0] = m_velocity[1];
    m_velocity[1] = tmp_velocity;
    size_t tmp_velocity_pitch = m_velocity_pitch[0];
    m_velocity_pitch[0] = m_velocity_pitch[1];
    m_velocity_pitch[1] = tmp_velocity_pitch;
}
void swap_velocity_host() {
    float2 *tmp_velocity = h_velocity[0];
    h_velocity[0] = h_velocity[1];
    h_velocity[1] = tmp_velocity;
    size_t tmp_velocity_pitch = h_velocity_pitch[0];
    h_velocity_pitch[0] = h_velocity_pitch[1];
    h_velocity_pitch[1] = tmp_velocity_pitch;
}
__global__ void swap_dye() {
    float3 *tmp_dye = m_dye[0];
    m_dye[0] = m_dye[1];
    m_dye[1] = tmp_dye;
    size_t tmp_dye_pitch = m_dye_pitch[0];
    m_dye_pitch[0] = m_dye_pitch[1];
    m_dye_pitch[1] = tmp_dye_pitch;
}
void swap_dye_host() {
    float3 *tmp_dye = h_dye[0];
    h_dye[0] = h_dye[1];
    h_dye[1] = tmp_dye;
    size_t tmp_dye_pitch = h_dye_pitch[0];
    h_dye_pitch[0] = h_dye_pitch[1];
    h_dye_pitch[1] = tmp_dye_pitch;
}


/**
 * @brief Advection Fragment Program
 *
 * @param result result
 * @param timestep
 * @param rdx reciprocal of the grid scale x
 * @param x Qty to be advected
 * @param u Velocity profile
 * @param width array width
 * @param height array height
 * @return __global__
 */
template <typename T>
__global__ void advect_kernel(T *result, size_t result_pitch,
                              float dt, float dissipation,
                              T *x, size_t x_pitch,
                              float2 *u, size_t u_pitch,
                              int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return;
    float2 pos = make_float2(i, j) - dt * u[j * u_pitch / sizeof(float2) + i];

    float2 floorPos = floorf(pos);
    float2 fractPos = pos - floorPos;

    int2 iFloorPos = make_int2(floorPos);

    if (iFloorPos.x < 0) {
        iFloorPos.x = 0;
        fractPos.x = 0;
    }
    if (iFloorPos.x >= width - 1) {
        iFloorPos.x = width - 1;
        fractPos.x = 1;
    }
    if (iFloorPos.y < 0) {
        iFloorPos.y = 0;
        fractPos.y = 0;
    }
    if (iFloorPos.y >= height - 1) {
        iFloorPos.y = height - 1;
        fractPos.y = 1;
    }

    T x00 = x[iFloorPos.y * x_pitch / sizeof(T) + iFloorPos.x];
    T x01 = x[iFloorPos.y * x_pitch / sizeof(T) + min(iFloorPos.x + 1, width - 1)];
    T x10 = x[min(iFloorPos.y + 1, height - 1) * x_pitch / sizeof(T) + iFloorPos.x];
    T x11 = x[min(iFloorPos.y + 1, height - 1) * x_pitch / sizeof(T) + min(iFloorPos.x + 1, width - 1)];

    T x0 = x00 * (1 - fractPos.x) + x01 * fractPos.x;
    T x1 = x10 * (1 - fractPos.x) + x11 * fractPos.x;
    T res = x0 * (1 - fractPos.y) + x1 * fractPos.y;

    result[j * result_pitch / sizeof(T) + i] = res / (1.0 + dt * dissipation);
}

/**
 * @brief The Jacobi Iteration Fragment Program Used to Solve Poisson Equations
 *
 * @param result result
 * @param x x vector (Ax = b)
 * @param b b vector (Ax = b)
 * @param alpha
 * @param rBeta reciprocal beta
 * @param width array width
 * @param height array height
 */
template <typename T>
__global__ void jacobi_kernel(T *result, size_t result_pitch,
                              T *x, size_t x_pitch,
                              T *b, size_t b_pitch,
                              float alpha, float rBeta, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return;

    if (i == 0 || j == 0 || i == width - 1 || j == height - 1) {
        result[j * result_pitch / sizeof(T) + i] = x[j * x_pitch / sizeof(T) + i];
        return;
    }

    // left, right, bottom, and top x samples
    T xL = x[j * x_pitch / sizeof(T) + i - 1];
    T xR = x[j * x_pitch / sizeof(T) + i + 1];
    T xB = x[(j - 1) * x_pitch / sizeof(T) + i];
    T xT = x[(j + 1) * x_pitch / sizeof(T) + i];

    // b sample, from center
    T bC = b[j * b_pitch / sizeof(T) + i];

    // evaluate Jacobi iteration
    T xNew = (xL + xR + xB + xT + alpha * bC) * rBeta;

    result[j * result_pitch / sizeof(T) + i] = xNew;
}

/**
 * @brief The Divergence Fragment Program
 *
 * @param result divergence
 * @param halfrdx 0.5 / gridscale
 * @param wX vector field (x component)
 * @param wY vector field (y component)
 * @param width array width
 * @param height array height
 * @return __global__
 */
__global__ void divergence_kernel(float *result, size_t result_pitch,
                                  float halfrdx,
                                  float2 *w, size_t w_pitch,
                                  int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return;

    float2 C = w[j * w_pitch / sizeof(float2) + i];
    
    float wL = w[j * w_pitch / sizeof(float2) + max(i - 1, 0)].x;
    float wR = w[j * w_pitch / sizeof(float2) + min(i + 1, width - 1)].x;
    float wB = w[max(j - 1, 0) * w_pitch / sizeof(float2) + i].y;
    float wT = w[min(j + 1, height - 1) * w_pitch / sizeof(float2) + i].y;

    if (i == 0) wL = -C.x;
    if (i == width - 1) wR = -C.x;
    if (j == 0) wB = -C.y;
    if (j == height - 1) wT = -C.y;

    float div = halfrdx * ((wR - wL) + (wT - wB));

    result[j * result_pitch / sizeof(float) + i] = div;
}

template <typename T>
__global__ void apply_force_kernel(T *result, size_t result_pitch,
                                   T *u, size_t u_pitch,
                                   float radius, float2 point, T F,
                                   int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return;

    if (i == 0 || j == 0 || i == width - 1 || j == height - 1) {
        result[j * result_pitch / sizeof(T) + i] = u[j * u_pitch / sizeof(T) + i];
        return;
    }

    T uv = u[j * u_pitch / sizeof(T) + i];
    float2 pos = make_float2(i, j);
    float dist = length(pos - point);
    T v_xy = F * expf(-(dist * dist) / radius);
    result[j * result_pitch / sizeof(T) + i] = uv + v_xy;
}


__global__ void gradient_kernel(float2 *result, size_t result_pitch,
                                float halfrdx,
                                float *p, size_t p_pitch,
                                float2 *w, size_t w_pitch,
                                int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return;

    float pL = p[j * p_pitch / sizeof(float) + max(i - 1, 0)];
    float pR = p[j * p_pitch / sizeof(float) + min(i + 1, width - 1)];
    float pB = p[max(j - 1, 0) * p_pitch / sizeof(float) + i];
    float pT = p[min(j + 1, height - 1) * p_pitch / sizeof(float) + i];

    float2 uNew = w[j * w_pitch / sizeof(float2) + i];
    uNew -= halfrdx * make_float2(pR - pL,  pT - pB);

    result[j * result_pitch / sizeof(float2) + i] = uNew;
}

__global__ void float3_to_uint8(uint8_t *result, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return;

    float2 val_vel = m_velocity[0][j * m_velocity_pitch[0] / sizeof(float2) + i];
    float val_pressure = m_pressure[0][j * m_pressure_pitch[0] / sizeof(float) + i];
    float val_divergence = m_divergence[j * m_divergence_pitch / sizeof(float) + i];
    float3 val_dye = m_dye[0][j * m_dye_pitch[0] / sizeof(float3) + i];

    val_vel = clamp(val_vel, -1.0f, 1.0f) * 0.5f + 0.5f;
    val_pressure = clamp(val_pressure, -1.0f, 1.0f) * 0.5f + 0.5f;
    val_divergence = clamp(val_divergence, -1.0f, 1.0f) * 0.5f + 0.5f;
    val_dye = clamp(val_dye, 0.0f, 1.0f);

    width *= 2;
    height *= 2;

    j = height - j - 1;  // OpenGL flip-y

    result[j * width * 4 + i * 4 + 0] = (uint8_t)(val_dye.x * 255);
    result[j * width * 4 + i * 4 + 1] = (uint8_t)(val_dye.y * 255);
    result[j * width * 4 + i * 4 + 2] = (uint8_t)(val_dye.z * 255);
    result[j * width * 4 + i * 4 + 3] = 255;

    i += width / 2;

    result[j * width * 4 + i * 4 + 0] = (uint8_t)(val_vel.x * 255);
    result[j * width * 4 + i * 4 + 1] = (uint8_t)(val_vel.y * 255);
    result[j * width * 4 + i * 4 + 2] = 255;
    result[j * width * 4 + i * 4 + 3] = 255;

    j -= height / 2;

    result[j * width * 4 + i * 4 + 0] = (uint8_t)(val_divergence * 255);
    result[j * width * 4 + i * 4 + 1] = 255;
    result[j * width * 4 + i * 4 + 2] = 255;
    result[j * width * 4 + i * 4 + 3] = 255;

    i -= width / 2;

    result[j * width * 4 + i * 4 + 0] = (uint8_t)(val_pressure * 255);
    result[j * width * 4 + i * 4 + 1] = 255;
    result[j * width * 4 + i * 4 + 2] = 255;
    result[j * width * 4 + i * 4 + 3] = 255;
}

void free_fluids() {
    checkCudaErrors(hipFree(h_velocity[0]));
    checkCudaErrors(hipFree(h_velocity[1]));
    checkCudaErrors(hipFree(h_pressure[0]));
    checkCudaErrors(hipFree(h_pressure[1]));
    checkCudaErrors(hipFree(h_divergence));
    checkCudaErrors(hipFree(h_dye[0]));
    checkCudaErrors(hipFree(h_dye[1]));
}

uint8_t *read_fully_file(const char *name, size_t *size) {
    FILE *f = fopen(name, "rb");
    if (!f) {
        fprintf(stderr, "Failed to open file %s\n", name);
        exit(1);
    }
    fseek(f, 0, SEEK_END);
    *size = ftell(f);
    fseek(f, 0, SEEK_SET);
    uint8_t *data = (uint8_t *)malloc(*size);
    if (!data) {
        fprintf(stderr, "Failed to allocate %zu bytes\n", *size);
        exit(1);
    }
    if (fread(data, 1, *size, f) != *size) {
        fprintf(stderr, "Failed to read %zu bytes\n", *size);
        exit(1);
    }
    fclose(f);
    return data;
}

void setup_fluids(int width, int height) {
    checkCudaErrors(hipMallocPitch(&h_velocity[0], &h_velocity_pitch[0], width * sizeof(float2), height));
    checkCudaErrors(hipMemset2D(h_velocity[0], h_velocity_pitch[0], 0, width * sizeof(float2), height));
    checkCudaErrors(hipMallocPitch(&h_velocity[1], &h_velocity_pitch[1], width * sizeof(float2), height));
    checkCudaErrors(hipMemset2D(h_velocity[1], h_velocity_pitch[1], 0, width * sizeof(float2), height));

    checkCudaErrors(hipMallocPitch(&h_pressure[0], &h_pressure_pitch[0], width * sizeof(float), height));
    checkCudaErrors(hipMemset2D(h_pressure[0], h_pressure_pitch[0], 0, width * sizeof(float), height));
    checkCudaErrors(hipMallocPitch(&h_pressure[1], &h_pressure_pitch[1], width * sizeof(float), height));
    checkCudaErrors(hipMemset2D(h_pressure[1], h_pressure_pitch[1], 0, width * sizeof(float), height));

    checkCudaErrors(hipMallocPitch(&h_divergence, &h_divergence_pitch, width * sizeof(float), height));
    checkCudaErrors(hipMemset2D(h_divergence, h_divergence_pitch, 0, width * sizeof(float), height));

    checkCudaErrors(hipMallocPitch(&h_dye[0], &h_dye_pitch[0], width * sizeof(float3), height));
    checkCudaErrors(hipMemset2D(h_dye[0], h_dye_pitch[0], 0, width * sizeof(float3), height));
    checkCudaErrors(hipMallocPitch(&h_dye[1], &h_dye_pitch[1], width * sizeof(float3), height));
    checkCudaErrors(hipMemset2D(h_dye[1], h_dye_pitch[1], 0, width * sizeof(float3), height));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_velocity), &h_velocity, sizeof(h_velocity)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_velocity_pitch), &h_velocity_pitch, sizeof(h_velocity_pitch)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_pressure), &h_pressure, sizeof(h_pressure)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_pressure_pitch), &h_pressure_pitch, sizeof(h_pressure_pitch)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_divergence), &h_divergence, sizeof(h_divergence)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_divergence_pitch), &h_divergence_pitch, sizeof(h_divergence_pitch)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_dye), &h_dye, sizeof(h_dye)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_dye_pitch), &h_dye_pitch, sizeof(h_dye_pitch)));

    size_t m_texture_size;
    uint8_t *m_texture = read_fully_file("winxp.raw", &m_texture_size);
    float3 *m_texture_float = (float3 *)malloc(h_dye_pitch[0] * height);
    for (int x = 0; x < width; x++) {
        for (int y = 0; y < height; y++) {
            int sx = x * 512 / width;
            int sy = y * 512 / height;
            int index = sy * 512 + sx;
            m_texture_float[y * h_dye_pitch[0] / sizeof(float3) + x] = make_float3(
                m_texture[index * 3 + 0] / 255.0f,
                m_texture[index * 3 + 1] / 255.0f,
                m_texture[index * 3 + 2] / 255.0f);
        }
    }
    free(m_texture);
    hipMemcpy(h_dye[0], m_texture_float, h_dye_pitch[0] * height, hipMemcpyHostToDevice);
    hipMemcpy(h_dye[1], m_texture_float, h_dye_pitch[1] * height, hipMemcpyHostToDevice);
    free(m_texture_float);
}

void apply_force(Controller *controller) {
    int width = controller->width;
    int height = controller->height;
    dim3 gridDim((width + 31) / 32, (height + 31) / 32);
    dim3 blockDim(32, 32);

    float dX = controller->deltaX * 10.0f;
    float dY = controller->deltaY * 10.0f;

    float radius = sqrtf(dX * dX + dY * dY);
    if (radius < 1.0f) {
        return;
    }
    apply_force_kernel<float2><<<gridDim, blockDim>>>(
        h_velocity[1], h_velocity_pitch[1],
        h_velocity[0], h_velocity_pitch[0],
        radius,
        make_float2(controller->mouseX, controller->mouseY),
        make_float2(dX, dY),
        width, height);

    swap_velocity<<<1, 1>>>();
    swap_velocity_host();

    // float3 color = hsv2rgb(rand() * 1.0f / RAND_MAX, 1.0f, 1.0f) * 0.15f;
    // printf("%f %f %f\n", color.x, color.y, color.z);

    apply_force_kernel<float3><<<gridDim, blockDim>>>(
        h_dye[1], h_dye_pitch[1],
        h_dye[0], h_dye_pitch[0],
        radius,
        make_float2(controller->mouseX, controller->mouseY),
        controller->currentColor * 0.01f,
        width, height);

    swap_dye<<<1, 1>>>();
    swap_dye_host();
}

void divergence(Controller *controller) {
    int width = controller->width;
    int height = controller->height;
    dim3 gridDim((width + 31) / 32, (height + 31) / 32);
    dim3 blockDim(32, 32);
    float halfrdx = 0.5f;
    divergence_kernel<<<gridDim, blockDim>>>(
        h_divergence, h_divergence_pitch,
        halfrdx,
        h_velocity[0], h_velocity_pitch[0],
        width, height);
}

void advect_velocity(Controller *controller, double timestep) {
    int width = controller->width;
    int height = controller->height;
    dim3 gridDim((width + 31) / 32, (height + 31) / 32);
    dim3 blockDim(32, 32);
    advect_kernel<float2><<<gridDim, blockDim>>>(
        h_velocity[1], h_velocity_pitch[1],
        timestep, 0.2f,
        h_velocity[0], h_velocity_pitch[0],
        h_velocity[0], h_velocity_pitch[0],
        width, height);

    swap_velocity<<<1, 1>>>();
    swap_velocity_host();
}

void advect_dye(Controller *controller, double timestep) {
    int width = controller->width;
    int height = controller->height;
    dim3 gridDim((width + 31) / 32, (height + 31) / 32);
    dim3 blockDim(32, 32);

    advect_kernel<float3><<<gridDim, blockDim>>>(
        h_dye[1], h_dye_pitch[1],
        timestep, 1.0f,
        h_dye[0], h_dye_pitch[0],
        h_velocity[0], h_velocity_pitch[0],
        width, height);

    swap_dye<<<1, 1>>>();
    swap_dye_host();
}

void computePressure(Controller *controller) {
    int width = controller->width;
    int height = controller->height;
    dim3 gridDim((width + 31) / 32, (height + 31) / 32);
    dim3 blockDim(32, 32);

    hipMemset2D(h_pressure[0], h_pressure_pitch[0], 0, width * sizeof(float), height);

    for (int i = 0; i < 5; i++) {
        jacobi_kernel<float><<<gridDim, blockDim>>>(
            h_pressure[1], h_pressure_pitch[1],
            h_pressure[0], h_pressure_pitch[0],
            h_divergence, h_divergence_pitch,
            -1.0f, 0.25f,
            width, height);
        jacobi_kernel<float><<<gridDim, blockDim>>>(
            h_pressure[0], h_pressure_pitch[0],
            h_pressure[1], h_pressure_pitch[1],
            h_divergence, h_divergence_pitch,
            -1.0f, 0.25f,
            width, height);
    }
}

void gradient(Controller *controller) {
    int width = controller->width;
    int height = controller->height;
    dim3 gridDim((width + 31) / 32, (height + 31) / 32);
    dim3 blockDim(32, 32);

    gradient_kernel<<<gridDim, blockDim>>>(
        h_velocity[1], h_velocity_pitch[1],
        1.0f,
        h_pressure[0], h_pressure_pitch[0],
        h_velocity[0], h_velocity_pitch[0],
        width, height);

    swap_velocity<<<1, 1>>>();
    swap_velocity_host();
}

void update_fluids(Controller *controller, double timestep) {
    if (controller->mouseButtons[GLFW_MOUSE_BUTTON_LEFT]) {
        apply_force(controller); // -> velocity
    }

    divergence(controller); // velocity -> divergence
    computePressure(controller); // divergence -> pressure
    gradient(controller); // velocity,pressure -> velocity

    advect_velocity(controller, timestep); // velocity -> velocity
    advect_dye(controller, timestep); // velocity,dye -> dye
}